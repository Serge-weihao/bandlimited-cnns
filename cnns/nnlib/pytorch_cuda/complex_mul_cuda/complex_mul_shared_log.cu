#include "hip/hip_runtime.h"
// #include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <iostream>
#include <cassert>

namespace {

/**
Complex multiplication of tensors using shared memory with barrier
synchronization and final summation across channels in the logarithmic manner.

Compute the element wise complex multiplication for each thread in the block and
write the result to the shared memory. Then synchronize the threads and in the
log based fashion sum up the results for each output pixel through its channels,
if they are present in the cache. The stride is the number of threads per block
times the I (the two float representation of complex numbers).
*/
template <typename scalar_t>
__global__ void complex_mul_cuda_kernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ y,
    scalar_t* __restrict__ out,
    const int N, const int F, const int H, const int W, const int C) {
    // The size of the shared memory cache should be twice the number of threads
    // per block as we store the real and imaginary part of the result.
    extern __shared__ float cache[];   // cache for the result of the complex multiplication

    const int I = 2; // the last dimension for the complex number
    const int plane_size = H * W;
    const int channel_size = C * I;
    const int image_size = plane_size * channel_size;  // size of the image from the batch
    // number of complex values in the input that we iterate through
    const int nr_values = H * W * C;

    const int n = blockIdx.x; // current index of an image/input map in the batch
    const int f = blockIdx.y; // current index of a filter from the filter bank
    const int block_size = blockDim.x;

    // After running all the threads in the block, we increment the thread
    // number by the block size.
    int thread_nr = threadIdx.x;

    // stride for the H*W map is equal to the number of threads declared in a block
    const int stride = block_size * I; // we need H*W threads per plane, each deals with I numbers

    const int n_idx = n * image_size;  // start index in the batch for this input map
    const int f_idx = f * image_size;  // start index in the bank for this filter

    // find index for the output
    const int output_fchannel_size = plane_size * I;
    const int no_idx = n * (F * output_fchannel_size); // output index for the batch data point
    const int fo_idx = f * output_fchannel_size;       // output index for the filter/channel

    // We go through each complex number one by one.
    // We linearize it and start from 0, move by #threads*I steps in outer loop.
    const int start_idx = threadIdx.x*I;

    // index in the input map
    int N_idx = n_idx + start_idx; // index across the first channel plane (in the input map n).
    const int last_N_idx = n_idx + image_size;  // last index for the starting position to compute the sum through each channel for this pixel

    // To prevent us from a deadlock, we have to always execute __syncthreads();
    // for all the threads in the block. Each thread has to do the same number of
    // iterations for any loop. To ensure that, we keep all threads running,
    // even though, some of them are really idle. We keep the loop running to
    // the multiple of the block size that is greater than the number of values
    // in the input map in total: C*H*W - this is a number of complex cells in the
    // input map.
    const int num_blocks = (nr_values + block_size - 1) / block_size;
    const int last_block_idx = n_idx + num_blocks * block_size * I;

    // Index in the filter (the filter is of exactly the same size and
    // dimensions as the input map.
    int F_idx = f_idx + start_idx;

    // index in the output, we compute cells on a flat plane (no channels).
    int base_O_idx = no_idx + fo_idx;

    // Cache (c) index;
    int thread_cidx = thread_nr * I;

    printf("N_idx:%d, last_block_idx:%d, last_N_idx:%d\n", N_idx, last_block_idx, last_N_idx);

    while (N_idx < last_block_idx)  {

        // Zero out caches.
        cache[thread_cidx] = 0;
        cache[thread_cidx + 1] = 0;

        if (N_idx < last_N_idx - 1) {
            scalar_t out_re = 0;
            scalar_t out_im = 0;

            scalar_t x_re = x[N_idx];
            scalar_t x_im = x[N_idx + 1];
            scalar_t y_re = y[F_idx];
            scalar_t y_im = y[F_idx + 1];
            single_mul(x_re, x_im, y_re, y_im, &out_re, &out_im);

            cache[thread_cidx] = out_re;
            cache[thread_cidx + 1] = out_im;
        }

        __syncthreads();  // Make the results visible to all threads.

        // Summed the pixels across channels.
        // It is of complexity O(logN). For each element in the output
        // map we add the computed pixels summed across channels.
        // This goes through all the channels present in the cache.

        sum_channels(cache, threadIdx.x, C);

        __syncthreads();
        // Write the output for the pixels summed (across channels).
        if (threadIdx.x % C == 0) {
            // Running index through the current XY plane in the output.
            // Assume 8 threads. Plane size HxW = 3x2 = 6 and 3 channels.
            // There are 6*3=18 complex multiplications for the output f-th plane.
            // The 0th, 1st, and 2nd threads should return 0 index in the output.
            // 3rd,4th,5th threads should return 1st index in the output.
            // The 20th index should return
            int run_O_idx = (thread_nr % (plane_size * C)) / C;

            const int O_idx = base_O_idx + run_O_idx*I;
            out[O_idx] += cache[thread_cidx];
            out[O_idx + 1] = cache[thread_cidx + 1];
        }

        N_idx += stride;
        F_idx += stride;
        thread_nr += block_size;

        // Make sure that all cache cells are zeroed out before moving on.
        // We need this as in the second part we access cache cells that do not
        // belong only to this thread.
        __syncthreads();
    }
}

template <typename scalar_t>
__device__ __forceinline__ void single_mul(
    scalar_t x_re,
    scalar_t x_im,
    scalar_t y_re,
    scalar_t y_im,
    scalar_t* out_re,
    scalar_t* out_im) {

    scalar_t uavc = x_re * (y_re + y_im);
    *out_re += uavc - (x_re + x_im) * y_im;
    *out_im += (x_im - x_re) * y_re + uavc;
}

template <typename scalar_t>
__device__ __forceinline__ void single_add(
    scalar_t x_re,
    scalar_t x_im,
    scalar_t y_re,
    scalar_t y_im,
    scalar_t* out_re,
    scalar_t* out_im) {

    *out_re += x_re + y_re;
    *out_im += x_im + y_im;
}

/**
Cache is with complex numbers. Sum the complex channels for a given pixel.

:param cache: an array of complex values to be summed for C consecutive complex
elements.
:param cache_index: the position of the thread in the cache.
:param C: number of channels.
*/
template <typename scalar_t>
__device__ __forceinline__
void sum_channels(scalar_t* cache, int cache_index, int C) {
    const int I = 2;  // complex number representation as 2 float numbers
    int c = C;  // C - number of all channels, c - still to be summed channels
    while (c != 0) {
        // printf("cache_index:%d, c:%d\n", cache_index, c);
        bool is_write = false;  // should we sum the values up with this thread?
        if (c % 2 == 0 || c == 1) {
            c /= 2;
            if (cache_index % C < c) {
                is_write = true;
            }
        } else {
            c = (c+1)/2;
            if (cache_index % C < c - 1) {
                is_write = true;
            }
        }
        if (is_write) {
            const int cache_index_I = cache_index*I;
            const int c_I = c*I;
            cache[cache_index_I] += cache[cache_index_I + c_I];
            cache[cache_index_I + 1] += cache[cache_index_I + c_I + 1];
        }
        __syncthreads();
        // printf("%d: %d, %d\n", cache_index, cache[cache_index*I], cache[cache_index*I+1]);
    }
}

template <typename scalar_t>
__global__ void test_sum_channels_device(scalar_t* cache, int C) {
    // printf("threadIdx.x: %d, C:%d\n", threadIdx.x, C);
    sum_channels(cache, threadIdx.x, C);
}

void test_sum_channels_host() {
    int *x, *y;
    const int I = 2;
    const int C = 2;
    const int W = 3;
    int size_input = W*C*I;

    // Allocate unified memory - accessible from cpu or gpu
    // hipMallocManaged(&x, size_input*sizeof(int));
    x = new int[size_input];
    y = new int[size_input];
    int *d_x;
    hipMalloc(&d_x, size_input*sizeof(int));

    for (int i = 0; i < size_input; ++i) {
        x[i] = i;
        y[i] = i;
    }

    printf("Initial numbers:\n");
    for (int w=0; w<W; ++w) {
        int w_idx = w*C*I;
        for (int c=0; c<C; ++c) {
           int c_idx = c*I;
           printf("(w:%d, c:%d): %d + %dj\n", w, c, x[w_idx + c_idx], x[w_idx+c_idx+1]);
        }
    }

    hipMemcpy(d_x, x, size_input*sizeof(int), hipMemcpyHostToDevice);

    const dim3 blocks(1);
    const int cuda_block_threads = 6;

    test_sum_channels_device<int><<<blocks, cuda_block_threads>>>(d_x, C);

    // Wait for GPU to finish before accessing on host.
    hipDeviceSynchronize();

    hipMemcpy(x, d_x, size_input*sizeof(int), hipMemcpyDeviceToHost);

    printf("Expected numbers for the output map (after summing up channels):\n");
    // Generate the expected output y - only for each channel.
    for (int w=0; w<W; ++w) {
        int w_idx = w*C*I;   // channels are on the last but one dimension
        printf("(w:%d): %d + %dj\n", w, y[w_idx], y[w_idx + 1]);
        for (int c = 1; c < C; ++c) {
            int c_idx = c*I;
            printf("(w:%d, c:%d): %d + %dj\n", w, c, y[w_idx + c_idx], y[w_idx + c_idx + 1]);
            y[w_idx] += y[w_idx + c_idx];
            y[w_idx + 1] += y[w_idx + c_idx + 1];
        }
        printf("expected: (w:%d): %d + %dj\n", w, y[w_idx], y[w_idx + 1]);
        printf("obtained: (w:%d): %d + %dj\n", w, x[w_idx], x[w_idx + 1]);
        assert (y[w_idx] == x[w_idx]);
        assert (y[w_idx + 1] == x[w_idx + 1]);
    }

    printf("Obtained numbers:\n");
    for (int w=0; w<W; ++w) {
        int w_idx = w*C*I;
        for (int c=0; c<C; ++c) {
           int c_idx = c*I;
           printf("(w:%d, c:%d): %d + %dj\n", w, c, x[w_idx + c_idx], x[w_idx+c_idx+1]);
        }
    }

//    for (int i=0; i < size_input; ++i) {
//        printf("%d: %d\n", i, x[i]);
//        // assert (expect[i] == x[i++]);
//    }

    hipFree(d_x);
    hipDeviceSynchronize();
    delete [] x;
    delete [] y;

    printf("finished test sum channels\n");
}

void test_sum_channels_host_big(int C, int W) {
    int *x, *y;
    const int I = 2;

    int size_input = W*C*I;

    // Allocate unified memory - accessible from cpu or gpu
    // hipMallocManaged(&x, size_input*sizeof(int));
    x = new int[size_input];
    y = new int[size_input];
    int *d_x;
    hipMalloc(&d_x, size_input*sizeof(int));

    for (int i = 0; i < size_input; ++i) {
        x[i] = i;
        y[i] = i;
    }

    printf("Initial numbers:\n");
    for (int w=0; w<W; ++w) {
        int w_idx = w*C*I;
        for (int c=0; c<C; ++c) {
           int c_idx = c*I;
           printf("(w:%d, c:%d): %d + %dj\n", w, c, x[w_idx + c_idx], x[w_idx+c_idx+1]);
        }
    }

    hipMemcpy(d_x, x, size_input*sizeof(int), hipMemcpyHostToDevice);

    const dim3 blocks(1);
    const int cuda_block_threads = W*C;

    test_sum_channels_device<int><<<blocks, cuda_block_threads>>>(d_x, C);

    // Wait for GPU to finish before accessing on host.
    hipDeviceSynchronize();

    hipMemcpy(x, d_x, size_input*sizeof(int), hipMemcpyDeviceToHost);

    printf("Expected numbers for the output map (after summing up channels):\n");
    // Generate the expected output y - only for each channel.
    for (int w=0; w<W; ++w) {
        int w_idx = w*C*I;   // channels are on the last but one dimension
        for (int c = 1; c < C; ++c) {
            int c_idx = c*I;
            y[w_idx] += y[w_idx + c_idx];
            y[w_idx + 1] += y[w_idx + c_idx + 1];
        }
        printf("expected: (w:%d): %d + %dj\n", w, y[w_idx], y[w_idx + 1]);
        printf("obtained: (w:%d): %d + %dj\n", w, x[w_idx], x[w_idx + 1]);
        assert (y[w_idx] == x[w_idx]);
        assert (y[w_idx + 1] == x[w_idx + 1]);
    }

    printf("Obtained numbers:\n");
    for (int w=0; w<W; ++w) {
        int w_idx = w*C*I;
        for (int c=0; c<C; ++c) {
           int c_idx = c*I;
           printf("(w:%d, c:%d): %d + %dj\n", w, c, x[w_idx + c_idx], x[w_idx+c_idx+1]);
        }
    }

    hipFree(d_x);
    hipDeviceSynchronize();
    delete [] x;
    delete [] y;

    printf("finished test sum channels\n");
}

} // namespace

//void complex_mul_shared_log_cuda(
//    at::Tensor x,
//    at::Tensor y,
//    at::Tensor out) {
//
//    const auto N = x.size(0);  // batch_size
//    const auto F = y.size(0);  // filter_bank_size
//    const auto H = x.size(1);  // height of the matrix
//    const auto W = x.size(2);  // width of the matrix
//    const auto C = x.size(3);  // number of channels
//
//    const auto x_blocks = N;
//    const auto y_blocks = F;
//    const dim3 blocks(x_blocks, y_blocks);
//
//    const int threads = int(1024/C) * C;
//
//    AT_DISPATCH_FLOATING_TYPES(x.type(), "complex_mul_cuda",
//    ([&] {
//        complex_mul_cuda_kernel<scalar_t><<<blocks, threads>>>(
//        x.data<scalar_t>(), y.data<scalar_t>(), out.data<scalar_t>(),
//        N, F, C, H, W);
//    }));
//}

//template <typename scalar_t>
//void complex_mul_stride_no_permute_cuda_pure(
//    at::Tensor x,
//    at::Tensor y,
//    at::Tensor out,
//    int threads = 1024) {
//
//    const auto N = x.size(0);  // batch_size
//    const auto F = y.size(0);  // filter_bank_size
//    const auto C = x.size(1);  // number of channels
//    const auto H = x.size(2);  // height of the matrix
//    const auto W = x.size(3);  // width of the matrix
//
//    const auto x_blocks = N;
//    const auto y_blocks = F;
//    const dim3 blocks(x_blocks, y_blocks);
//
//    // Run kernel on the GPU
//    complex_mul_cuda_kernel<scalar_t><<<blocks, 1024>>>(
//        x.data<scalar_t>(), y.data<scalar_t>(), out.data<scalar_t>(),
//        N, F, C, H, W);
//}

/**
Uncomment the pytorch related stuff.

Compile:
ady@skr-compute1:/tmp/pycharm_project_154/cnns/nnlib/pytorch_cuda/complex_mul_cuda$ nvcc complex_mul_kernel_stride_no_permute.cu -o complex_mul_profile.out
ady@skr-compute1:/tmp/pycharm_project_154/cnns/nnlib/pytorch_cuda/complex_mul_cuda$ nvprof ./complex_mul_profile.out

nvidia

/usr/local/cuda/bin/nvcc -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/torch/csrc/api/include -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/TH -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/THC -I/usr/local/cuda/include -I/local/ady/anaconda3/include/python3.6m -c complex_mul_kernel.cu -o complex_mul_kernel_stride_no_permute.out -std=c++11
nvcc -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/torch/csrc/api/include -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/TH -I/local/ady/anaconda3/lib/python3.6/site-packages/torch/lib/include/THC -I/usr/local/cuda/include -I/local/ady/anaconda3/include/python3.6m complex_mul_kernel_stride_no_permute.cu -o complex_mul_kernel_stride_no_permute.out -std=c++11
Segmentation fault
*/

void test_multiply() {
    int N = 1;
    int F = 1;
    int H = 3;
    int W = 2;
    int C = 3;
    int I = 2;
    int size_input = N * H * W * C * I;
    int size_filter = F * H * W * C * I;
    int size_output = N * F * H * W * I;
    int cuda_block_threads = int(16/3) * 3;

    // auto dims = {128, 32, 16, 8, 2};
    //    at::Tensor x = at::randn({128, 32, 16, 8, 2});
    //    at::Tensor y = at::randn({128, 32, 16, 8, 2});
    //    at::Tensor out = at::zeros({128, 32, 16, 8, 2});
    float *x, *y, * out;

    // Allocate unified memory - accessible from cpu or gpu
    hipMallocManaged(&x, size_input*sizeof(float));
    hipMallocManaged(&y, size_filter*sizeof(float));
    hipMallocManaged(&out, size_output*sizeof(float));

    for (int j=0; j<H; ++j) {
        for (int i=0; i<W; ++i) {
            for (int c=0; c<C; ++c) {
                const int index = (j*W*C+i*C+c)*2;
                x[index] = index;
                x[index + 1] = index + 1;
                y[index] = 4;
                y[index + 1] = 2;
            }
        }
    }

    for (int i=0; i<H*W*2; i+=2) {
        printf("%p %d: %f, %f, %f, %f\n", x, i, x[i], x[i+1], y[i], y[i+1]);
    }

    const dim3 blocks(N, F);

    complex_mul_cuda_kernel<float><<<blocks, cuda_block_threads,
        cuda_block_threads*2>>>(x, y, out, N, F, H, W, C);

    for (int i=0; i<H*W*C; i+=2) {
        printf("%d: %f, %f\n", i, out[i], out[i+1]);
    }

    hipFree(x);
    hipFree(y);
    hipFree(out);

    printf("finished computation\n");
}

void test_sum_channels_suit() {
    test_sum_channels_host();
    test_sum_channels_host_big(/*C=*/1, /*W=*/7);
    test_sum_channels_host_big(/*C=*/1, /*W=*/1024);
    test_sum_channels_host_big(/*C=*/3, /*W=*/300);
    test_sum_channels_host_big(/*C=*/1, /*W=*/19);
    test_sum_channels_host_big(/*C=*/4, /*W=*/4);
    test_sum_channels_host_big(/*C=*/16, /*W=*/6);
    test_sum_channels_host_big(/*C=*/32, /*W=*/32);
    test_sum_channels_host_big(/*C=*/3, /*W=*/300);
}

int main(void)
{
    test_sum_channels_suit();
    // test_multiply();

    return 0;
}