#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <torch/torch.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

// The size of block of threads.
const int blocksize = 1024;

//__device__ double atomicAdd(double *address, double val) {
//    unsigned long long int *address_as_ull =
//            (unsigned long long int *) address;
//    unsigned long long int old = *address_as_ull, assumed;
//    do {
//        assumed = old;
//        old = atomicCAS(address_as_ull, assumed,
//                        __double_as_longlong(val +
//                                             __longlong_as_double(assumed)));
//    } while (assumed != old);
//    return __longlong_as_double(old);
//}

template<typename scalar_t>
__global__ void plus_reduce_cuda_kernel(
        scalar_t *input,
        const int64_t __restrict__ input_size,
        scalar_t *total_sum) {
    const unsigned int tid = threadIdx.x;
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Each block loads its elements into shared memory, padding with 0 if
    // input_size is not a multiple of blocksize.

    // Sharing the data for the block of threads.
    __shared__
    scalar_t x[blocksize];

    // For the last block we might have to add the zero elements at the end.
    x[tid] = (index < input_size) ? input[index] : 0;

    // Synchronize threads (this is a barrier) to update the cache - the same
    // coherent view of all the threads on the GPU.
    __syncthreads();

    // Every thread now holds 1 input value in x[].

    // Build summation tree over the elements (the blockDim.x is a power of 2).
    for (int s = blockDim.x / 2; s > 0; s = s / 2) {
        // Every thread holds sum of blocksize/s elements.
        if (tid < s) x[tid] += x[tid + s];
        __syncthreads();
    }

    // Thread 0 now holds the sum of all the input values to this block. Have it
    // add that sum to the running total.
    if (tid == 0) atomicAdd(total_sum, x[0]);
}

at::Tensor plus_reduce_cuda(at::Tensor input) {
    // at::Scalar total_sum = at::Scalar()
    at::Tensor total_sum = at::zeros({1});
    const int64_t input_size = input.size(0);

    const dim3 blocks((input_size + blocksize - 1) / blocksize, blocksize);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "plus_reduce_cuda", ([&] {
        plus_reduce_cuda_kernel<scalar_t> << < blocks, blocksize >> >
                                                       (input.data<scalar_t>(), input_size, total_sum.data<scalar_t>());
    }));

    return total_sum;
}

std::vector <at::Tensor> conv_cuda_forward(
        at::Tensor input,
        at::Tensor filter,
        at::Tensor bias,
        at::Tensor padding,
        at::Tensor index_back) {
    return {};
}

std::vector <at::Tensor> conv_cuda_backward(
        at::Tensor dout,
        at::Tensor xfft,
        at::Tensor yfft,
        at::Tensor W,
        at::Tensor WW,
        at::Tensor fft_size) {
    return {};
}